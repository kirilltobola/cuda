﻿
#include "hip/hip_runtime.h"


#include <iostream>
#include <cstdio>
#include <stdio.h>
#include <thread>

#include <chrono>

#define N 10000000

__global__ void addKernel(int * c, const int * a, const int * b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

void add(int * c, const int * a, const int * b)
{
	for (int i = 0; i < N; i++) {
		c[i] = a[i] + b[i];
	}
}

void call_add(int * c, const int * a, const int * b)
{
	auto s = std::chrono::steady_clock::now();
	add(c, a, b);
	auto e = std::chrono::steady_clock::now();

	std::cout << "CPU elapsed time: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(e - s).count()
		<< " ms" << std::endl;

	// Print results
	int sum_ = 0;
	for (int i = 0; i < N; i++) {
		sum_ += c[i];
	}
	std::cout << "sum = " << sum_ << std::endl;
}

int main()
{
	int * vec1 = new int[N];
	int * vec2 = new int[N];
	int * res = new int[N];

	// Init vectors
	for (int i = 0; i < N; i++) {
		vec1[i] = 1;
		vec2[i] = 0;
	}

	// CPU
	call_add(res, vec1, vec2);

	// Init vectors
	for (int i = 0; i < N; i++) {
		vec1[i] = 1;
		vec2[i] = 0;
	}

	// Alloc memory on gpu
	int * dev_vec1 = nullptr;
	int * dev_vec2 = nullptr;
	int * dev_res = nullptr;
	hipMalloc((void**)&dev_vec1, N * sizeof(int));
	hipMalloc((void**)&dev_vec2, N * sizeof(int));
	hipMalloc((void**)&dev_res, N * sizeof(int));

	// From Host to Device
	hipMemcpy(dev_vec1, vec1, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_vec2, vec2, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_res, res, N * sizeof(int), hipMemcpyHostToDevice);

	// Measure time
	float time;
	hipEvent_t	start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Launch a kernel on the GPU with one block; N threads.
	addKernel <<<1, N>>> (dev_res, dev_vec1, dev_vec2);
	hipDeviceSynchronize();

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("GPU elapsed time: %.5f ms \n", time);

	// From Device to Host
	hipMemcpy(res, dev_res, N * sizeof(int), hipMemcpyDeviceToHost);

	// Print results
	int sum = 0;
	for (int i = 0; i < N; i++) {
		sum += res[i];
		// std::cout << res[i] << std::endl;
	}
	std::cout << "sum = " << sum << std::endl;

	hipFree(dev_vec1);
	hipFree(dev_vec2);
	hipFree(dev_res);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipDeviceReset();

    return 0;
}
